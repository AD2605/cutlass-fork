#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0
#  include "cutlass/util/cublas_wrappers.hpp"
#endif
#include "cutlass/util/helper_cuda.hpp"

using namespace cute;

using TileShape = Shape<_128, _128, _32>;

using TiledMma = TiledMMA<
        MMA_Atom<SM80_16x8x8_F32TF32TF32F32_TN>,
        Layout<Shape<_2,_2,_1>, Stride<_2, _1, _1>>, // 2x2x1 thread group
        Layout<Shape<_1,_2,_1>>>;                    // 1x2x1 value group for 16x16x8 and LDSM

// Smem
using SmemLayoutAtomA = decltype(
composition(Swizzle<2,3,2>{},
            Layout<Shape <_32, _8>,
                    Stride< _1,_32>>{}));
using SmemCopyAtomA = Copy_Atom<UniversalCopy<tfloat32_t>, tfloat32_t>;
// Gmem
using GmemTiledCopyA = decltype(
make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, tfloat32_t>{},
                Layout<Shape <_16, _8>,
                        Stride< _1,_16>>{},
                Layout<Shape < _4, _1>>{}));

// Smem
using SmemLayoutAtomB = decltype(
composition(Swizzle<2,3,2>{},
            Layout<Shape <_32, _8>,
                    Stride< _1,_32>>{}));
using SmemCopyAtomB = Copy_Atom<UniversalCopy<tfloat32_t>, tfloat32_t>;
// Gmem
using GmemTiledCopyB = decltype(
make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, tfloat32_t>{},
                Layout<Shape <_16, _8>,
                        Stride< _1,_16>>{},
                Layout<Shape < _4, _1>>{}));

using Stages = Int<3>;

using SmemLayoutA = decltype(tile_to_shape(
        SmemLayoutAtomA{},
        make_shape(shape<0>(TileShape{}), shape<2>(TileShape{}), Stages{})));
using SmemLayoutB = decltype(tile_to_shape(
        SmemLayoutAtomB{},
        make_shape(shape<1>(TileShape{}), shape<2>(TileShape{}), Stages{})));

struct float2tf32 {
    CUTE_HOST_DEVICE
    tfloat32_t operator()(float&& arg) const {
      uint32_t storage = reinterpret_cast<uint32_t &>(arg);
      bool mantissa_bit = ((storage & (1 << 13)) != 0);
      bool round_bit = ((storage & (1 << 12)) != 0);
      bool sticky_bit = ((storage & ((1 << 12) - 1)) != 0);

      if ((round_bit && sticky_bit) || (round_bit && mantissa_bit)) {
        storage += uint32_t(1 << 13);
      }

      return tfloat32_t::bitcast(storage);
    }
};

template <class MShape, class NShape, class KShape,
          class TA, class AStride,
          class TB, class BStride,
          class TC, class CStride,
          class Alpha, class Beta>
__global__
void
gemm_device(MShape M, NShape N, KShape K,
            TA const* A, AStride dA,
            TB const* B, BStride dB,
            TC      * C, CStride dC,
            Alpha alpha, Beta beta) {
  using namespace cute;
  using X = Underscore;

  // Shared memory buffers
  extern __shared__ tfloat32_t smem[];
  tfloat32_t* smemA = smem;
  tfloat32_t* smemB = smem + cosize_v<SmemLayoutA>;
  auto sA = make_tensor(make_smem_ptr(smemA), SmemLayoutA{});               // (BLK_M,BLK_K)
  auto sB = make_tensor(make_smem_ptr(smemB), SmemLayoutB{});               // (BLK_N,BLK_K)

  // Represent the full tensors
  auto mA = make_tensor(make_gmem_ptr(A), make_shape(M,K), dA);      // (M,K)
  auto mB = make_tensor(make_gmem_ptr(B), make_shape(N,K), dB);      // (N,K)
  auto mC = make_tensor(make_gmem_ptr(C), make_shape(M,N), dC);      // (M,N)

  // Get the appropriate blocks for this thread block --
  // potential for thread block locality
  auto blk_shape = TileShape{};// (BLK_M,BLK_N,BLK_K)
  // Compute m_coord, n_coord, and l_coord with their post-tiled shapes
  auto m_coord = idx2crd(int(blockIdx.x), shape<0>(blk_shape));
  auto n_coord = idx2crd(int(blockIdx.y), shape<1>(blk_shape));
  auto blk_coord = make_coord(m_coord, n_coord, _);            // (m,n,k)

  auto gA = local_tile(mA, blk_shape, blk_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  auto gB = local_tile(mB, blk_shape, blk_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  auto gC = local_tile(mC, blk_shape, blk_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  //
  // Partition the copying of A and B tiles across the threads
  //

  GmemTiledCopyA gmem_tiled_copy_A;
  GmemTiledCopyB gmem_tiled_copy_B;
  auto gmem_thr_copy_A = gmem_tiled_copy_A.get_slice(threadIdx.x);
  auto gmem_thr_copy_B = gmem_tiled_copy_B.get_slice(threadIdx.x);

  Tensor tAgA = gmem_thr_copy_A.partition_S(gA);                             // (ACPY,ACPY_M,ACPY_K,k)
  Tensor tAsA = gmem_thr_copy_A.partition_D(sA);                             // (ACPY,ACPY_M,ACPY_K,PIPE)
  Tensor tBgB = gmem_thr_copy_B.partition_S(gB);                             // (BCPY,BCPY_N,BCPY_K,k)
  Tensor tBsB = gmem_thr_copy_B.partition_D(sB);                             // (BCPY,BCPY_N,BCPY_K,PIPE)

  auto k_tile_iter  = make_coord_iterator(shape<2>(gA));
  int  k_tile_count = size<2>(gA);

  for (int k_pipe = 0; k_pipe < Stages{}-1; ++k_pipe) {
    copy(gmem_tiled_copy_A, tAgA(_,_,_,*k_tile_iter), tAsA(_,_,_,k_pipe));
    copy(gmem_tiled_copy_B, tBgB(_,_,_,*k_tile_iter), tBsB(_,_,_,k_pipe));
    cp_async_fence();
    ++k_tile_iter;
    --k_tile_count;
  }

  //
  // Define C accumulators and A/B partitioning
  //

  TiledMma tiled_mma;
  auto thr_mma = tiled_mma.get_thread_slice(threadIdx.x);
  Tensor tCrA  = thr_mma.partition_fragment_A(sA(_,_,0));                    // (MMA,MMA_M,MMA_K)
  Tensor tCrB  = thr_mma.partition_fragment_B(sB(_,_,0));                    // (MMA,MMA_N,MMA_K)
  Tensor tCgC = thr_mma.partition_C(gC);

  auto smem_tiled_copy_A   = make_tiled_copy_A(SmemCopyAtomA{}, tiled_mma);
  auto smem_thr_copy_A     = smem_tiled_copy_A.get_thread_slice(threadIdx.x);
  Tensor tCsA           = smem_thr_copy_A.partition_S(sA);                   // (CPY,CPY_M,CPY_K,PIPE)
  Tensor tCrA_copy_view = smem_thr_copy_A.retile_D(tCrA);
  CUTE_STATIC_ASSERT_V(size<1>(tCsA) == size<1>(tCrA_copy_view));            // CPY_M
  CUTE_STATIC_ASSERT_V(size<2>(tCsA) == size<2>(tCrA_copy_view));            // CPY_K

  auto smem_tiled_copy_B = make_tiled_copy_B(SmemCopyAtomB{}, tiled_mma);
  auto smem_thr_copy_B   = smem_tiled_copy_B.get_thread_slice(threadIdx.x);
  Tensor tCsB              = smem_thr_copy_B.partition_S(sB);                // (CPY,CPY_N,CPY_K,PIPE)
  Tensor tCrB_copy_view    = smem_thr_copy_B.retile_D(tCrB);
  CUTE_STATIC_ASSERT_V(size<1>(tCsB) == size<1>(tCrB_copy_view));            // CPY_N
  CUTE_STATIC_ASSERT_V(size<2>(tCsB) == size<2>(tCrB_copy_view));            // CPY_K

  // Allocate the accumulators -- same size as the projected data
  Tensor tCrC = partition_fragment_C(tiled_mma, take<0,2>(blk_shape)); // (MMA,MMA_M,MMA_N)

  // Clear the accumulators
  clear(tCrC);

  // Current pipe index in smem to read from
  int smem_pipe_read  = 0;
  // Current pipe index in smem to write to
  int smem_pipe_write = Stages{}-1;

  Tensor tCsA_p = tCsA(_,_,_,smem_pipe_read);
  Tensor tCsB_p = tCsB(_,_,_,smem_pipe_read);

  // Size of the register pipeline
  auto K_BLOCK_MAX = size<2>(tCrA);

  // PREFETCH register pipeline
  if (K_BLOCK_MAX > 1) {
    // Wait until our first prefetched tile is loaded in
    cp_async_wait<Stages{}-2>();
    __syncthreads();

    // Prefetch the first rmem from the first k-tile
    copy(smem_tiled_copy_A, tCsA_p(_,_,Int<0>{}), tCrA_copy_view(_,_,Int<0>{}));
    copy(smem_tiled_copy_B, tCsB_p(_,_,Int<0>{}), tCrB_copy_view(_,_,Int<0>{}));
  }

  CUTE_NO_UNROLL
  for ( ; k_tile_count > -(Stages{}-1); --k_tile_count)
  {
    // Pipeline the outer products with a static for loop.
    //
    // Note, the for_each() function is required here to ensure `k_block` is of type Int<x>.
    CUTE_UNROLL
    for_each(make_int_sequence<K_BLOCK_MAX>{}, [&] (auto k_block) {
      if (k_block == K_BLOCK_MAX - 1) {
        // Slice the smem_pipe_read smem
        tCsA_p = tCsA(_, _, _, smem_pipe_read);
        tCsB_p = tCsB(_, _, _, smem_pipe_read);

        // Commit the smem for smem_pipe_read
        cp_async_wait<Stages{} - 2>();
        __syncthreads();
      }

      // Load A, B shmem->regs for k_block+1
      auto k_block_next = (k_block + Int<1>{}) % K_BLOCK_MAX;  // static
      copy(smem_tiled_copy_A, tCsA_p(_, _, k_block_next), tCrA_copy_view(_, _, k_block_next));
      copy(smem_tiled_copy_B, tCsB_p(_, _, k_block_next), tCrB_copy_view(_, _, k_block_next));
      // Copy gmem to smem before computing gemm on each k-pipe
      if (k_block == 0) {
        if (k_tile_count > 0) {
          copy(gmem_tiled_copy_A, tAgA(_, _, _, *k_tile_iter), tAsA(_, _, _, smem_pipe_write));
          copy(gmem_tiled_copy_B, tBgB(_, _, _, *k_tile_iter), tBsB(_, _, _, smem_pipe_write));
          cp_async_fence();
          ++k_tile_iter;
        }
        // Advance the pipe -- Doing it here accounts for K_BLOCK_MAX = 1 (no rmem pipe)
        smem_pipe_write = smem_pipe_read;
        ++smem_pipe_read;
        smem_pipe_read = (smem_pipe_read == Stages{}) ? 0 : smem_pipe_read;
      }

      // Transform before compute
      cute::transform(tCrA(_, _, k_block), identity());
      cute::transform(tCrB(_, _, k_block), identity());
      // Thread-level register gemm for k_block
      cute::gemm(tiled_mma, tCrC, tCrA(_, _, k_block), tCrB(_, _, k_block), tCrC);
    });
  }

  //
  // Epilogue
  //

  // Represent the full output tensor
  Tensor mD_mnl = make_tensor(make_gmem_ptr(C), make_shape(M,N), dC);                 // (m,n,l)
  Tensor gD_mnl = local_tile(mD_mnl, blk_shape, make_coord(_,_,_), Step<_1,_1, X>{});    // (BLK_M,BLK_N,m,n,l)

  // Slice to get the tile this CTA is responsible for
  Tensor gD = gD_mnl(_,_,m_coord,n_coord);                                                 // (BLK_M,BLK_N)

  // Partition source and destination tiles to match the accumulator partitioning
  Tensor tCgD = thr_mma.partition_C(gD);                                       // (VEC,THR_M,THR_N)

  // Make an identity coordinate tensor for predicating our output MN tile
  auto cD = make_identity_tensor(make_shape(unwrap(shape<0>(gD)), unwrap(shape<1>(gD))));
  Tensor tCcD = thr_mma.partition_C(cD);

  // source is not needed, avoid load
  CUTE_UNROLL
  for (int i = 0; i < size(tCrC); ++i) {
    tCgD(i) = tCrC(i);
  }
}


template <typename TA, typename TB, typename TC,
          typename Alpha, typename Beta>
void
gemm(int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);

  // Define strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);
  auto dB = make_stride(Int<1>{}, ldB);
  auto dC = make_stride(Int<1>{}, ldC);

  // Define block sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 32>{};

  // Define the block layouts (static)
  auto sC = make_layout(make_shape(bM,bN));

  auto sA = tile_to_shape(SmemLayoutAtomA{}, make_shape(bM,bK));
  auto sB = tile_to_shape(SmemLayoutAtomB{}, make_shape(bN,bK));

  dim3 dimBlock(128, 1, 1);
  dim3 dimGrid(ceil_div(size(M), size(bM)),
               ceil_div(size(N), size(bN)));

  int smem_size = (cosize_v<SmemLayoutA> + cosize_v<SmemLayoutA>) * sizeof(tfloat32_t);

  if (smem_size >= (48 << 10)) {
    auto result = hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_device<int), int, int, TA, decltype(dA), TB, decltype(dB),
                                               TC, decltype(dC), Alpha, Beta>,
                                       hipFuncAttributeMaxDynamicSharedMemorySize,
                                       smem_size);

    if (result != hipSuccess) {
      printf("hipFuncSetAttribute error\n");
      return;
    }
  }

  gemm_device<<< dimGrid, dimBlock, smem_size, stream >>>(
          M,  N,  K,
          A, dA,
          B, dB,
          C, dC,
          alpha, beta);
}

void test_gemm(int m, int n, int k)
{
  cute::device_init(0);

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;

  using TA = float;
  using TB = float;
  using TC = float;
  using TI = float;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( j % 11 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( j % 11 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  TI alpha = 1.0;
  TI beta  = 0.0;

  double tflops = (2.0*m*n*k) * 1e-12;

  const int timing_iterations = 100;
  GPU_Clock timer;

#if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0
  //
  // cuBLas
  //

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Run once
  d_C = h_C;
  blam::cublas::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                     m, n, k,
                     &alpha,
                     d_A.data().get(), m,
                     d_B.data().get(), n,
                     &beta,
                     d_C.data().get(), m);
  CUTE_CHECK_LAST();

  thrust::host_vector<TC> cublas_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    blam::cublas::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                       m, n, k,
                       &alpha,
                       d_A.data().get(), m,
                       d_B.data().get(), n,
                       &beta,
                       d_C.data().get(), m);
  }
  double cublas_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUBLAS_GEMM:   [%4.3f]TFlop/s  (%6.4f)ms\n", tflops / cublas_time, cublas_time*1000);

#else

  std::cout << "Verification by comparison with cuBLAS is disabled, "
    "either because the CMake option CUTLASS_ENABLE_CUBLAS "
    "was explicitly set to OFF, or because CMake could not find cuBLAS.  "
    "If you would like to enable verification with cuBLAS, "
    "please set the CMake option CUTLASS_ENABLE_CUBLAS to ON, "
    "rerun CMake, and recompile this example.\n";

#endif // CUTLASS_ENABLE_CUBLAS

  //
  // CuTe
  //

  // Run once (and check)
  d_C = h_C;
  gemm(m, n, k,
       alpha,
       d_A.data().get(), m,
       d_B.data().get(), n,
       beta,
       d_C.data().get(), m);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(m, n, k,
         alpha,
         d_A.data().get(), m,
         d_B.data().get(), n,
         beta,
         d_C.data().get(), m);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%4.3f]TFlop/s  (%6.4f)ms\n", tflops / cute_time, cute_time*1000);

#if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0
  printf("Empirical Perf: %.1f%%\n", (cublas_time / cute_time) * 100);

  auto host_matrix_to_const_column_major_cute_tensor =
    [](const auto& X, int num_rows, int num_cols, int LDX) {
      const auto shape = cute::Shape<int, int>{num_rows, num_cols};
      const auto strides = cute::Stride<int, int>{1, LDX};
      return cute::make_tensor(X.data(), cute::make_layout(shape, strides));
    };

  const auto A_view = host_matrix_to_const_column_major_cute_tensor(h_A, m, k, m);
  // B^T is k x n, so B is n x k.
  const auto B_view = host_matrix_to_const_column_major_cute_tensor(h_B, n, k, n);
  const auto C_computed_view = host_matrix_to_const_column_major_cute_tensor(cute_result, m, n, m);
  const auto C_expected_view = host_matrix_to_const_column_major_cute_tensor(cublas_result, m, n, m);
  print_matrix_multiply_mollified_relative_error("float", A_view, B_view, C_computed_view, C_expected_view);

#endif // CUTLASS_ENABLE_CUBLAS
}

int main(int argc, char** argv)
{
  int m = 5120;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 5120;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 4096;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  test_gemm(m, n, k);

  return 0;
}
